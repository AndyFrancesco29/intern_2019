#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <math.h>
#include <ComplexRandNum.cuh>



__global__ void getRand() 
{
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	int N = 3;
	CudaComplex *temp = (CudaComplex*)malloc(N * sizeof(CudaComplex));
	getDeviceRand(id,N,temp);
	CudaComplex sum=temp[0]+temp[1]+temp[2];
	sum.display();
}


int main()
{
	//hipFree(devStates);
	//hipFree(N2);
	////hipFree(N3);
	//return 0;

	/*int N = 10000;
	CudaComplex *result;
	hipMallocManaged(&result, N * sizeof(CudaComplex));
	getRandomNumber(result, N, 128);
	result[1000].display();
	return 0;*/

	getRand << <2, 8 >> > ();
	return 0;
}