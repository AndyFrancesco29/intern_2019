#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <math.h>


__device__ float generate_rand(hiprandState *state, int i) {
	hiprandState localstate = state[i];
	float random = hiprand_uniform(&localstate);
	state[i] = localstate;
	return random;
}

__global__ void setup_kernel(hiprandState *state, unsigned long int seed, int* count) {
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	//printf("%d\n", i);
	hiprand_init(seed, i, 2, &state[i]);
	count[i] = i;
}

__global__ void kernel(float *N, hiprandState *state)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	float k = generate_rand(state, i);
	N[i] = k;
	//printf("%f\n", k);

}

int main()
{
	int N = 40000;
	hiprandState* devStates;
	hipMalloc((void**)&devStates, N * sizeof(hiprandState));

	int *count;
	hipMallocManaged(&count, sizeof(int)*N);
	hipMemset(count, 0, sizeof(int)*N);

	float *N2;
	hipMallocManaged(&N2, N * sizeof(float));
	//float *N3;
	//hipMalloc((void**)&N3, sizeof(float)*N);

	hipMemset(N2, 0, sizeof(float)*N);

	int threadsPerBlock = 256;
	int numBlock = (N + threadsPerBlock - 1) / threadsPerBlock;
	//printf("%d\n", gridsize);

	setup_kernel << <numBlock, threadsPerBlock >> > (devStates, time(NULL), count);
	hipDeviceSynchronize();

	kernel << <numBlock, threadsPerBlock >> > (N2, devStates);
	hipDeviceSynchronize();

	//hipMemcpy(N2,N3,sizeof(float)*N,hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
	{
		//if(N2[i]==0)
		//	printf("%d\n",i);
		printf("%f\n", N2[i]);
		//if (count[i] != i)
		//	printf("%d\n", i);
	}

	hipFree(N2);
	//hipFree(N3);
	return 0;
}