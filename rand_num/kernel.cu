#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <math.h>


__device__ float generate_rand(hiprandState *state, int i) {
	hiprandState localstate = state[i];
	float random = hiprand_uniform(&localstate);
	state[i] = localstate;
	return random;
}

__global__ void setup_kernel(hiprandState *state, unsigned long int seed) {
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	//printf("%d\n", i);
	hiprand_init(seed, i, 2, &state[i]);
	
}

__global__ void kernel(float *N, hiprandState *state,int num_state, int num_rand)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	int d = i;
	for (int i; i < num_rand; i+=num_state)
	{
		float k = generate_rand(state, d);
		N[i] = k;
	}
	//printf("%f\n", k);

}

int main()
{
	int N = 1<<20;
	int state_n = 256;
	hiprandState* devStates;
	hipMallocManaged((void**)&devStates, state_n * sizeof(hiprandState));

	float *N2;
	hipMallocManaged(&N2, N * sizeof(float));
	//float *N3;
	//hipMalloc((void**)&N3, sizeof(float)*N);

	hipMemset(N2, 0, sizeof(float)*N);

	int threadsPerBlock = 256;
	int numBlock = (N + threadsPerBlock - 1) / threadsPerBlock;
	int stateBlock = (state_n + threadsPerBlock - 1) / threadsPerBlock;
	//printf("%d\n", gridsize);

	setup_kernel << <stateBlock, threadsPerBlock >> > (devStates, time(NULL));
	hipDeviceSynchronize();

	kernel << <stateBlock, threadsPerBlock >> > (N2, devStates,state_n,N);
	hipDeviceSynchronize();

	//hipMemcpy(N2,N3,sizeof(float)*N,hipMemcpyDeviceToHost);

	//for (int i = 0; i < N; i++)
	//{
	//	if(N2[i]==N2[100])
	//		printf("%d\n",i);
	//	printf("%f\n", N2[i]);
	//}

	hipFree(devStates);
	hipFree(N2);
	//hipFree(N3);
	return 0;
}